

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <sys/time.h>

#define DataType double
#define TPB 32

// Compute C = A * B
__global__ void gemm(DataType *A, DataType *B, DataType *C, int numARows,
                      int numAColumns, int numBRows, int numBColumns){
  //@@ Insert code to implement matrix multiplication here
  int col = blockIdx.x * blockDim.x + threadIdx.x;
  int row = blockIdx.y * blockDim.y + threadIdx.y;

  if (row < numARows && col < numBColumns) {
    DataType value = 0;
    for (int k = 0; k < numAColumns; k++) {
      value += A[row * numAColumns + k] * B[k * numBColumns + col];
    }
    C[row * numBColumns + col] = value;
  }

}

int main(int argc, char **argv) {
  
  DataType *hostA; // The A matrix
  DataType *hostB; // The B matrix
  DataType *hostC; // The output C matrix
  DataType *resultRef; // The reference result
  DataType *deviceA;
  DataType *deviceB;
  DataType *deviceC;
  int numARows;    // number of rows in the matrix A
  int numAColumns; // number of columns in the matrix A
  int numBRows;    // number of rows in the matrix B
  int numBColumns; // number of columns in the matrix B
  int numCRows;
  int numCColumns;

  //@@ Insert code below to read in numARows, numAColumns, numBColumns from args
  numARows =  atoi(argv[1]);
  numAColumns = atoi(argv[2]);
  numBRows = numAColumns;
  numBColumns = atoi(argv[3]);
  numCRows = numARows;
  numCColumns = numBColumns;

  printf("Input matrix dim (%d x %d) (%d x %d) (%d x %d)\n", numARows, numAColumns, numBRows, numBColumns, numCRows, numCColumns);
  
  //@@ Insert code below to allocate Host memory for input and output
  hostA = (DataType*)malloc(numARows * numAColumns * sizeof(DataType));
  hostB = (DataType*)malloc(numBRows * numBColumns * sizeof(DataType));
  hostC = (DataType*)malloc(numCRows * numCColumns * sizeof(DataType));
  resultRef = (DataType*)malloc(numCRows * numCColumns * sizeof(DataType));
  
  //@@ Insert code below to initialize hostA and hostB to random numbers, and create reference result in CPU
  for (int i = 0; i < numARows * numAColumns; i++) {
      hostA[i] = rand()/(DataType)RAND_MAX;
  }

  for (int i = 0; i < numBRows * numBColumns; i++) {
      hostB[i] = rand()/(DataType)RAND_MAX;
  }

  for (int i = 0; i < numARows; i++) {
    for (int j = 0; j < numBColumns; j++) {
      DataType value = 0;
      for (int k = 0; k < numAColumns; k++) {
        value += hostA[i * numAColumns + k] * hostB[k * numBColumns + j];
      }
      resultRef[i * numBColumns + j] = value;
    }
  }

  //@@ Insert code below to allocate GPU memory here

  hipMalloc(&deviceA, numARows * numAColumns * sizeof(DataType));
  hipMalloc(&deviceB, numBRows * numBColumns * sizeof(DataType));
  hipMalloc(&deviceC, numCRows * numCColumns * sizeof(DataType));


  //@@ Insert code to below to Copy memory to the GPU here

  hipMemcpy(deviceA, hostA, numARows * numAColumns * sizeof(DataType), hipMemcpyHostToDevice);
  hipMemcpy(deviceB, hostB, numBRows * numBColumns * sizeof(DataType), hipMemcpyHostToDevice);


  //@@ Initialize the grid and block dimensions here
  dim3 blockSize(TPB, TPB);
  dim3 gridSize((numARows + TPB - 1) / TPB, (numBColumns + TPB - 1) / TPB);


  //@@ Launch the GPU Kernel here
  gemm<<<gridSize, blockSize>>>(deviceA, deviceB, deviceC, numARows, numAColumns, numBRows, numBColumns);
  hipDeviceSynchronize();


  //@@ Copy the GPU memory back to the CPU here
  hipMemcpy(hostC, deviceC, numCRows * numCColumns * sizeof(DataType), hipMemcpyDeviceToHost);


  //@@ Insert code below to compare the output with the reference

  bool match = true;
  for (int i = 0; i < numCRows * numCColumns; i++) {
    if (fabs(hostC[i] - resultRef[i]) > 1e-6) {
      match = false;
      printf("Mismatch at index %d: hostOutput[%d] = %f, resultRef[%d] = %f\n", i, i, hostC[i], i, resultRef[i]);
      break;
    }
  }
  if (match) {
    printf("Test PASSED\n");
  } else {
    printf("Test FAILED\n");
  }

  //@@ Free the GPU memory here

  hipFree(deviceA);
  hipFree(deviceB);
  hipFree(deviceC);

  //@@ Free the CPU memory here
  free(hostA);
  free(hostB);
  free(hostC);
  free(resultRef);

  return 0;
}
